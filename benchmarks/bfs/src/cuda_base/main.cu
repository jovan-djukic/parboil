#include "hip/hip_runtime.h"
/***********************************************************************************
	Implementing Breadth first search on CUDA using algorithm given in DAC'10
	paper "An Effective GPU Implementation of Breadth-First Search"

	Copyright (c) 2010 University of Illinois at Urbana-Champaign.
	All rights reserved.

	Permission to use, copy, modify and distribute this software and its documentation for
	educational purpose is hereby granted without fee, provided that the above copyright
	notice and this permission notice appear in all copies of this software and that you do
	not sell the software.

	THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR
	OTHERWISE.

	Author: Lijiuan Luo (lluo3@uiuc.edu)
	Revised for Parboil 2 Benchmark Suite by: Geng Daniel Liu (gengliu2@illinois.edu)
 ************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <parboil.h>
#include <deque>
#include <iostream>

#include "config.h"

typedef int2 Node;
typedef int2 Edge;

#include "kernel.cu"

const int h_top = 1;
const int zero	= 0;

int main ( int argc, char** argv ) {
	struct pb_TimerSet timers;
	pb_InitializeTimerSet ( &timers );

	struct pb_Parameters* parameters = pb_ReadParameters ( &argc, argv );
	if ( ( parameters->inpFiles[0] == NULL ) || ( parameters->inpFiles[1] != NULL ) ) {
	fprintf ( stderr, "Expecting one input filename\n" );
	exit ( -1 );
	}

	pb_SwitchToTimer ( &timers, pb_TimerID_IO );

	// Read graph from a file
	FILE *file = fopen ( parameters->inpFiles[0], "r" );
	if ( !file ) {
		printf ( "Error Reading graph file\n" );
		return 0;
	}

	// Read the number of nodes in the graph
	int num_of_nodes = 0;
	fscanf ( file, "%d", &num_of_nodes );

	// Allocate host memory
    Node* h_graph_nodes = ( Node* ) malloc ( sizeof ( Node ) * num_of_nodes );
    int*  color         = ( int* ) malloc ( sizeof ( int ) * num_of_nodes );
	// Initalize the memory
	for ( int i = 0; i < num_of_nodes; ++i ) {
    	int	 start, edgeno;
		fscanf ( file, "%d %d", &start, &edgeno );

		h_graph_nodes[i].x = start;
		h_graph_nodes[i].y = edgeno;

	    color[i] = WHITE;
    }
	// Read the source node and the number of edges in graph from the file
    int source       = 0;
    int num_of_edges = 0;
    fscanf ( file, "%d", &source );
	fscanf ( file, "%d", &num_of_edges );

	Edge* h_graph_edges = ( Edge* ) malloc ( sizeof ( Edge ) * num_of_edges );
		for ( int i = 0; i < num_of_edges; ++i ) {
		int	 id, cost;
		fscanf ( file, "%d", &id );
		fscanf ( file, "%d", &cost );

		h_graph_edges[i].x = id;
		h_graph_edges[i].y = cost;
	}

	if ( file ) {
		fclose ( file );
	}

	// Allocate memory for the result on host side
	int* h_cost = ( int* ) malloc ( sizeof ( int ) * num_of_nodes );
	for ( int i = 0; i < num_of_nodes; ++i ) {
		h_cost[i] = INFINITY;
	}
	h_cost[source] = 0;

	pb_SwitchToTimer ( &timers, pb_TimerID_COPY );

	// Copy the Node list to device memory
	Node* d_graph_nodes;
	hipMalloc ( ( void** ) &d_graph_nodes, sizeof ( Node ) * num_of_nodes );
	hipMemcpy ( d_graph_nodes, h_graph_nodes, sizeof ( Node ) * num_of_nodes, hipMemcpyHostToDevice );

	// Copy the Edge List to device Memory
	Edge* d_graph_edges;
	hipMalloc ( ( void** ) &d_graph_edges, sizeof ( Edge ) * num_of_edges );
	hipMemcpy ( d_graph_edges, h_graph_edges, sizeof ( Edge ) * num_of_edges, hipMemcpyHostToDevice );

	int* d_color;
	hipMalloc ( ( void** ) &d_color, sizeof ( int ) * num_of_nodes );
	hipMemcpy ( d_color, color, sizeof ( int ) * num_of_nodes, hipMemcpyHostToDevice );

	int* d_cost;
	hipMalloc ( ( void** ) &d_cost, sizeof ( int ) * num_of_nodes );
	hipMemcpy ( d_cost, h_cost, sizeof ( int ) * num_of_nodes, hipMemcpyHostToDevice );

	int* d_q1;
	int* d_q2;
	hipMalloc ( ( void** ) &d_q1, sizeof ( int ) * num_of_nodes );
	hipMalloc ( ( void** ) &d_q2, sizeof ( int ) * num_of_nodes );

	int* tail;
	hipMalloc ( ( void** ) &tail, sizeof ( int ) );

	int* front_cost_d;
	hipMalloc ( ( void** ) &front_cost_d, sizeof ( int ) );

	// Bind the texture memory with global memory
	hipBindTexture ( 0, g_graph_node_ref, d_graph_nodes, sizeof ( Node ) * num_of_nodes );
	hipBindTexture ( 0, g_graph_edge_ref, d_graph_edges, sizeof ( Edge ) * num_of_edges );

	printf ( "Starting GPU kernel\n" );
	hipDeviceSynchronize ( );
	pb_SwitchToTimer ( &timers, pb_TimerID_KERNEL );

	hipMemcpy ( tail, &h_top, sizeof ( int ), hipMemcpyHostToDevice );
	hipMemcpy ( &d_cost[source], &zero, sizeof ( int ), hipMemcpyHostToDevice );

	hipMemcpy ( &d_q1[0], &source, sizeof ( int ), hipMemcpyHostToDevice );
	int num_t; // number of threads
	int k = 0; // BFS level index

	do {
		hipMemcpy ( &num_t, tail, sizeof ( int ), hipMemcpyDeviceToHost );
		hipMemcpy ( tail, &zero, sizeof ( int ), hipMemcpyHostToDevice );

		// frontier is empty
		if ( num_t == 0 ) { 			
			break;
		}

	    int num_of_blocks            = 1;
	    int num_of_threads_per_block = num_t;

	    if ( num_of_threads_per_block < NUM_BIN ) {
			num_of_threads_per_block = NUM_BIN;
		}

		if ( num_t > MAX_THREADS_PER_BLOCK ) {
	        num_of_blocks            = ( int ) ceil ( num_t / ( double ) MAX_THREADS_PER_BLOCK );
	        num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
	    }

		// will call "BFS_in_GPU_kernel"
		if ( num_of_blocks == 1 ) {
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
		}

		// will call "BFS_kernel_multi_blk_inGPU"
		if ( num_of_blocks > 1 && num_of_blocks <= NUM_SM ) {
			num_of_blocks = NUM_SM;
		}

		dim3 grid ( num_of_blocks, 1, 1 );
		dim3 threads ( num_of_threads_per_block, 1, 1 );

		if ( k % 2 == 0 ) {
			BFS_kernel<<<grid, threads>>> ( 
				d_q1,
				d_q2,
				d_graph_nodes,
				d_graph_edges,
				d_color,
				d_cost,
				num_t,
				tail,
				GRAY0,
				k 
			);
		} else {
			BFS_kernel<<<grid, threads>>> ( 
				d_q2,
				d_q1,
				d_graph_nodes,
				d_graph_edges,
				d_color,
				d_cost,
				num_t,
				tail,
				GRAY1,
				k 
			);
		}

		k++;
	} while ( 1 );

	hipDeviceSynchronize ( );
	pb_SwitchToTimer ( &timers, pb_TimerID_COPY );
	printf ( "GPU kernel done\n" );

	// Copy result from device to host
	hipMemcpy ( h_cost, d_cost, sizeof ( int ) * num_of_nodes, hipMemcpyDeviceToHost );
	hipMemcpy ( color, d_color, sizeof ( int ) * num_of_nodes, hipMemcpyDeviceToHost );
	hipUnbindTexture ( g_graph_node_ref );
	hipUnbindTexture ( g_graph_edge_ref );

	hipFree ( d_graph_nodes );
	hipFree ( d_graph_edges );
	hipFree ( d_color );
	hipFree ( d_cost );
	hipFree ( tail );
	hipFree ( front_cost_d );

	// Store the result into a file
	pb_SwitchToTimer ( &timers, pb_TimerID_IO );

	file = fopen ( parameters->outFile, "w" );

	fprintf ( file, "%d\n", num_of_nodes );
	for ( int i = 0; i < num_of_nodes; ++i ) {
		fprintf ( file, "%d %d\n", i, h_cost[i] );
	}
	fclose ( file );

	// cleanup memory
	free ( h_graph_nodes );
	free ( h_graph_edges );
	free ( color );
	free ( h_cost );
	pb_SwitchToTimer ( &timers, pb_TimerID_NONE );
	pb_PrintTimerSet ( &timers );
	pb_FreeParameters ( parameters );
	return 0;
}
