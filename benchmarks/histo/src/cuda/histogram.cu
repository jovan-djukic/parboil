
#include <hip/hip_runtime.h>
#define UINT8_MAX 255

__global__ void histogramGPU ( unsigned int* image, unsigned int imageLength, unsigned int* histogram ) {
    int tx = threadIdx.x; 
    int bx = blockIdx.x;

    int index = ( bx * blockDim.x ) + tx;

    if ( index < imageLength ) {
        unsigned int value = image[index];

        atomicAdd ( &histogram[value], 1 );
    }
}

__global__ void saturate ( unsigned int *intHistogram, unsigned int histogramLength ) {
    int tx = threadIdx.x; 
    int bx = blockIdx.x;
    int index = ( bx * blockDim.x ) + tx;

    if ( index < histogramLength ) {
        if ( intHistogram[index] > UINT8_MAX ) {
            intHistogram[index] = UINT8_MAX;
        }     
    }
}