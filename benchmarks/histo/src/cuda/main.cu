#include "hip/hip_runtime.h"
#include <parboil.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "util.h"

#include <hip/hip_runtime.h>

#define MAX_THREADS_PER_BLOCK 1024

__global__ void histogramGPU ( unsigned int* data, unsigned int imageLength, unsigned int* histogram );
__global__ void saturate ( unsigned int *intHistogram, unsigned int imageLength );

int main ( int argc, char *argv[] ) {
	struct pb_Parameters *parameters = pb_ReadParameters ( &argc, argv );

	printf ( "Base implementation of histogramming.\n" );
	printf ( "Maintained by Nady Obeid <obeid1@ece.uiuc.edu>\n" );

	if ( !parameters ) {
		return -1;
	}

	if ( !parameters->inpFiles[0] ) {
		fputs ( "Input file expected\n", stderr );
		return -1;
	}

	int numberOfIterations = 0;
	if ( argc >= 2 ) {
		numberOfIterations = atoi ( argv[1] );
	} else {
		fputs ( "Expected at least one command line argument\n", stderr );
		return -1;
	}

	struct pb_TimerSet timers;
	pb_InitializeTimerSet ( &timers );

	char *inputStr = "Input";
	char *outputStr = "Output";

	pb_AddSubTimer ( &timers, inputStr, pb_TimerID_IO );
	pb_AddSubTimer ( &timers, outputStr, pb_TimerID_IO );

	pb_SwitchToSubTimer ( &timers, inputStr, pb_TimerID_IO );

	unsigned int img_width = 0;
	unsigned int img_height = 0;
	unsigned int histo_width = 0;
	unsigned int histo_height = 0;

	FILE *file = fopen ( parameters->inpFiles[0], "rb" );

	int result = 0;

	result += fread ( &img_width, sizeof(unsigned int), 1, file );
	result += fread ( &img_height, sizeof(unsigned int), 1, file );
	result += fread ( &histo_width, sizeof(unsigned int), 1, file );
	result += fread ( &histo_height, sizeof(unsigned int), 1, file );

	if ( result != 4 ) {
		fputs ( "Error reading input and output dimensions from file\n", stderr );
		return -1;
	}

	unsigned int *image = ( unsigned int* ) malloc ( img_width * img_height * sizeof ( unsigned int ) );
	unsigned int *histogram = ( unsigned int* ) calloc ( histo_width * histo_height, sizeof ( unsigned int ) );

	pb_SwitchToSubTimer ( &timers, "Input", pb_TimerID_IO );

	result = fread ( image, sizeof(unsigned int), img_width * img_height, file );

	fclose ( file );

	if ( result != img_width * img_height ) {
		fputs ( "Error reading input array from file\n", stderr );
		return -1;
	}

	unsigned int *deviceImage = NULL;
	unsigned int *intHistogram = NULL;

	hipMalloc ( ( void** ) &deviceImage, img_width * img_height * sizeof ( unsigned int ) );
	hipMalloc ( ( void** ) &intHistogram, histo_width * histo_height * sizeof ( unsigned int ) );

	hipMemcpy ( deviceImage, image, img_width * img_height * sizeof ( unsigned int ), hipMemcpyHostToDevice );

	pb_SwitchToTimer ( &timers, pb_TimerID_COMPUTE );

	for ( int iteration = 0; iteration < numberOfIterations; ++iteration ) {
		unsigned int imageLength = img_width * img_height;
		unsigned int histogramLength = histo_width * histo_height;

		hipMemset ( intHistogram, 0, histogramLength * sizeof ( unsigned int ) );

		dim3 imageBlockDimensions ( MAX_THREADS_PER_BLOCK, 1, 1 );
		int imageBlocks = ( imageLength + MAX_THREADS_PER_BLOCK - 1 ) / MAX_THREADS_PER_BLOCK;
		dim3 imageGridDimensions ( imageBlocks, 1, 1 );
		histogramGPU<<<imageBlockDimensions, imageGridDimensions>>> ( deviceImage, imageLength, intHistogram );

		dim3 histogramBlockDimensions ( MAX_THREADS_PER_BLOCK, 1, 1 );
		int histogramBlocks = ( histogramLength + MAX_THREADS_PER_BLOCK - 1 ) / MAX_THREADS_PER_BLOCK;
		dim3 histogramGridDimensions ( histogramBlocks, 1, 1 );
		saturate<<<histogramBlockDimensions, histogramGridDimensions>>> ( intHistogram, histogramLength );
	}

	//  pb_SwitchToTimer(&timers, pb_TimerID_IO);
	pb_SwitchToSubTimer ( &timers, outputStr, pb_TimerID_IO );

	hipMemcpy ( histogram, intHistogram, histo_width * histo_height * sizeof ( unsigned int ), hipMemcpyDeviceToHost );

	hipFree ( deviceImage );
	hipFree ( intHistogram );

	if ( parameters->outFile ) {
		dump_histo_img ( histogram, histo_height, histo_width, parameters->outFile );
	}

	pb_SwitchToTimer ( &timers, pb_TimerID_COMPUTE );

	free ( image );
	free ( histogram );

	pb_SwitchToTimer ( &timers, pb_TimerID_NONE );

	printf ( "\n" );
	pb_PrintTimerSet ( &timers );
	pb_FreeParameters ( parameters );

	return 0;
}
