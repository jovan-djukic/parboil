#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <parboil.h>

#include "UDTypes.h"
#include "CUDA_kernels.h"

#define PI 3.14159265

/************************************************************ 
 * This function reads the parameters from the file provided
 * as a comman line argument.
 ************************************************************/
void setParameters ( FILE *file, Parameters *p ) {
    fscanf ( file, "aquisition.numsamples=%d\n", &p->numSamples );
    fscanf ( file, "aquisition.kmax=%f %f %f\n", &p->kMax[0], &p->kMax[1], &p->kMax[2] );
    fscanf ( file, "aquisition.matrixSize=%d %d %d\n", &p->aquisitionMatrixSize[0], &p->aquisitionMatrixSize[1], &p->aquisitionMatrixSize[2] );
    fscanf ( file, "reconstruction.matrixSize=%d %d %d\n", &p->reconstructionMatrixSize[0], &p->reconstructionMatrixSize[1], &p->reconstructionMatrixSize[2] );
    fscanf ( file, "gridding.matrixSize=%d %d %d\n", &p->gridSize[0], &p->gridSize[1], &p->gridSize[2] );
    fscanf ( file, "gridding.oversampling=%f\n", &p->oversample );
    fscanf ( file, "kernel.width=%f\n", &p->kernelWidth );
    fscanf ( file, "kernel.useLUT=%d\n", &p->useLUT );

    printf ( "  Number of samples = %d\n", p->numSamples );
    printf ( "  Grid Size = %dx%dx%d\n", p->gridSize[0], p->gridSize[1], p->gridSize[2] );
    printf ( "  Input Matrix Size = %dx%dx%d\n", p->aquisitionMatrixSize[0], p->aquisitionMatrixSize[1], p->aquisitionMatrixSize[2] );
    printf ( "  Recon Matrix Size = %dx%dx%d\n", p->reconstructionMatrixSize[0], p->reconstructionMatrixSize[1], p->reconstructionMatrixSize[2] );
    printf ( "  Kernel Width = %f\n", p->kernelWidth );
    printf ( "  KMax = %.2f %.2f %.2f\n", p->kMax[0], p->kMax[1], p->kMax[2] );
    printf ( "  Oversampling = %f\n", p->oversample );
    printf ( "  GPU Binsize = %d\n", p->binsize );
    printf ( "  Use LUT = %s\n", (p->useLUT ) ? "Yes" : "No" );
}

/************************************************************ 
 * This function reads the sample point data from the kspace
 * and klocation files (and sdc file if provided) into the
 * sample array.
 * Returns the number of samples read successfully.
 ************************************************************/
unsigned int readSampleData ( Parameters params, FILE *uksdata_f, ReconstructionSample *samples ) {
    int count = 0;
    for ( unsigned int i = 0; i < params.numSamples; ++i ) {
        if ( feof ( uksdata_f ) ) {
            break;
        }

        fread ( ( (void*) &samples[i] ), sizeof ( ReconstructionSample ), 1, uksdata_f );
        count++;
    }

    float kScale[3];
    kScale[0] = ( (float) params.aquisitionMatrixSize[0] ) / ( ( (float) params.reconstructionMatrixSize[0] ) * ( (float) params.kMax[0] ) );
    kScale[1] = ( (float) params.aquisitionMatrixSize[1] ) / ( ( (float) params.reconstructionMatrixSize[1] ) * ( (float) params.kMax[1] ) );
    kScale[2] = ( (float) params.aquisitionMatrixSize[2] ) / ( ( (float) params.reconstructionMatrixSize[2] ) * ( (float) params.kMax[2] ) );

    int size_x = params.gridSize[0];
    int size_y = params.gridSize[1];
    int size_z = params.gridSize[2];

    float ax = ( kScale[0] * ( size_x - 1 ) ) / 2.0;
    float bx = (float) ( size_x - 1 ) / 2.0;

    float ay = ( kScale[1] * ( size_y - 1 ) ) / 2.0;
    float by = (float) ( size_y - 1 ) / 2.0;

    float az = ( kScale[2] * ( size_z - 1 ) ) / 2.0;
    float bz = (float) ( size_z - 1 ) / 2.0;

    for ( int n = 0; n < count; n++ ) {
        samples[n].kX = floor ( ( samples[n].kX * ax ) + bx );
        samples[n].kY = floor ( ( samples[n].kY * ay ) + by );
        samples[n].kZ = floor ( ( samples[n].kZ * az ) + bz );
    }

    return count;
}

float kernel_value_CPU ( float v ) {
    const float z = v * v;

    // polynomials taken from http://ccrma.stanford.edu/CCRMA/Courses/422/projects/kbd/kbdwindow.cpp
    float num = (  z * ( z * ( z * ( z * ( z * ( z * ( z * ( z * ( z * ( z * ( z * ( z * ( z *
                (  z * 0.210580722890567e-22f + 0.380715242345326e-19f ) +
                0.479440257548300e-16f ) + 0.435125971262668e-13f ) + 0.300931127112960e-10f ) +
                0.160224679395361e-7f ) + 0.654858370096785e-5f ) + 0.202591084143397e-2f ) +
                0.463076284721000e0f ) + 0.754337328948189e2f ) + 0.830792541809429e4f ) +
                0.571661130563785e6f ) + 0.216415572361227e8f ) + 0.356644482244025e9f ) +
                0.144048298227235e10f );

    float den = ( z * ( z * ( z - 0.307646912682801e4f ) + 0.347626332405882e7f ) - 0.144048298227235e10f );

    float rValue = -num / den;

    return rValue;
}

void calculateLUT ( float beta, float width, float **LUT, unsigned int *sizeLUT ) {
    const float cutoff2 = ( width * width ) / 4.0;

    if ( width > 0 ) {
        // compute size of LUT based on kernel width
        unsigned int size = (unsigned int) ( 10000 * width );

        // allocate memory
        (*LUT) = (float*) malloc ( size * sizeof ( float ) );

        for ( unsigned int k = 0; k < size; ++k ) {
            // compute value to evaluate kernel at
            // v in the range 0:(_width/2)^2
            float v = ( ((float) k) / ((float) size) ) * cutoff2;

            // compute kernel value and store
            (*LUT)[k] = kernel_value_CPU ( beta * sqrt ( 1.0 - ( v / cutoff2 ) ) );
        }

        (*sizeLUT) = size;
    }
}


int main ( int argc, char *argv[] ) {
    struct pb_Parameters *prms;
    struct pb_TimerSet timers;

    prms = pb_ReadParameters ( &argc, argv );
    pb_InitializeTimerSet ( &timers );

    pb_SwitchToTimer ( &timers, pb_TimerID_NONE );

    char uksdata[250];
    Parameters params;

    FILE *uksfile_f = NULL;
    FILE *uksdata_f = NULL;

    strcpy ( uksdata, prms->inpFiles[0] );
    strcat ( uksdata, ".data" );

    uksfile_f = fopen ( prms->inpFiles[0], "r" );
    if ( uksfile_f == NULL ) {
        printf ( "ERROR: Could not open %s\n", prms->inpFiles[0] );
        exit ( 1 );
    }

    printf ( "\nReading parameters\n" );

    if ( argc >= 2 ) {
        params.binsize = atoi ( argv[1] );
    } else { //default binsize value;
        params.binsize = 128;
    }

    setParameters ( uksfile_f, &params );

    pb_SwitchToTimer ( &timers, pb_TimerID_IO );

    ReconstructionSample *samples = (ReconstructionSample*) malloc ( params.numSamples * sizeof ( ReconstructionSample ) ); //Input Data
    float *LUT;                                                                                                       //use look-up table for faster execution on CPU (intermediate data)
    unsigned int sizeLUT;                                                                                             //set in the function calculateLUT (intermediate data)

    int gridNumElems = params.gridSize[0] * params.gridSize[1] * params.gridSize[2];

    cmplx *gridData = (cmplx*) calloc ( gridNumElems, sizeof ( cmplx ) );      //Output Data
    float *sampleDensity = (float*) calloc ( gridNumElems, sizeof ( float ) ); //Output Data

    if ( samples == NULL ) {
        printf ( "ERROR: Unable to allocate memory for input data\n" );
        exit ( 1 );
    }

    if ( sampleDensity == NULL || gridData == NULL ) {
        printf ( "ERROR: Unable to allocate memory for output data\n" );
        exit ( 1 );
    }

    uksdata_f = fopen ( uksdata, "rb" );

    if ( uksdata_f == NULL ) {
        printf ( "ERROR: Could not open data file\n" );
        exit ( 1 );
    }

    printf ( "Reading input data from files\n" );

    unsigned int n = readSampleData ( params, uksdata_f, samples );
    fclose ( uksdata_f );

    if ( params.useLUT ) {
        printf ( "Generating Look-Up Table\n" );
        float beta = PI * sqrt ( 4 * params.kernelWidth * params.kernelWidth / ( params.oversample * params.oversample ) * ( params.oversample - .5 ) * ( params.oversample - .5 ) - .8 );
        calculateLUT ( beta, params.kernelWidth, &LUT, &sizeLUT );
    }

    pb_SwitchToTimer ( &timers, pb_TimerID_COMPUTE );

    gridding_Gold ( n, params, samples, LUT, sizeLUT, gridData, sampleDensity );

    pb_SwitchToTimer ( &timers, pb_TimerID_IO );

    int passed = 1;

    FILE *outfile;
    if ( !( outfile = fopen ( prms->outFile, "w" ) ) ) {
        printf ( "Cannot open output file!\n" );
    } else {
        fwrite ( &passed, sizeof ( int ), 1, outfile );
        fclose ( outfile );
    }

    pb_SwitchToTimer ( &timers, pb_TimerID_NONE );

    if ( params.useLUT ) {
        free ( LUT );
    }
    
    free ( samples );
    free ( gridData );
    free ( sampleDensity );

    printf ( "\n" );
    pb_PrintTimerSet ( &timers );
    pb_FreeParameters ( prms );

    return 0;
}
